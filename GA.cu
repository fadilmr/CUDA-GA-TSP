#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


// Number of individuals in the population
const int populationSize = 2048;

// Number of genes in each individual
const int numCities = 10;

__constant__ float distances[numCities][numCities];

// CUDA kernel for parallel fitness evaluation
__global__ void calculateFitness(int* population, float* fitness) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < populationSize) {
        float tempFitness = 0.0f;
        for (int i = 0; i < numCities - 1; ++i) {
            tempFitness += distances[population[tid * numCities + i]][population[tid * numCities + i + 1]];
        }
        tempFitness += distances[population[tid * numCities + numCities - 1]][population[tid * numCities]];

        fitness[tid] = 1.0f / tempFitness;

        tid += blockDim.x * gridDim.x;
    }
}

__device__ int selectParent(float* fitness, hiprandState* state) {
    float totalFitness = 0.0f;
    for (int i = 0; i < populationSize; ++i) {
        totalFitness += fitness[i];
    }

    float randomFitness = totalFitness * hiprand_uniform(state);
    float accumulatedFitness = 0.0f;
    for (int i = 0; i < populationSize; ++i) {
        accumulatedFitness += fitness[i];
        if (accumulatedFitness >= randomFitness) {
            return i;
        }
    }
    return -1;
}

__global__ void crossoverAndMutation(int* population, float* fitness, hiprandState* states) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < populationSize) {
        hiprandState state = states[tid];

        int parent1 = selectParent(fitness, &state);
        int parent2 = selectParent(fitness, &state);

        int crossoverPoint = hiprand(&state) % numCities;
        for (int i = crossoverPoint; i < numCities; ++i) {
            int temp = population[parent1 * numCities + i];
            population[parent1 * numCities + i] = population[parent2 * numCities + i];
            population[parent2 * numCities + i] = temp;
        }

        int mutationPoint = hiprand(&state) % numCities;
        population[parent1 * numCities + mutationPoint] = hiprand(&state) % numCities;
        population[parent2 * numCities + mutationPoint] = hiprand(&state) % numCities;

        states[tid] = state;
    }
}

int main() {
    // Host arrays
    float* h_population = new float[populationSize * numCities];
    float* h_fitness = new float[populationSize];

    // Configure and launch CUDA kernel
    int blockSize = 512;
    int numBlocks = (populationSize + blockSize - 1) / blockSize;

    // Number of generations
    int numGenerations = 2000;

    // Initialize population randomly
    srand(static_cast<unsigned>(time(nullptr)));
    for (int i = 0; i < populationSize * numCities; ++i) {
        h_population[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Device arrays
    float* d_population;
    float* d_fitness;
    hipMalloc((void**)&d_population, sizeof(float) * populationSize * numCities);
    hipMalloc((void**)&d_fitness, sizeof(float) * populationSize);

    // Copy population from host to device
    hipMemcpy(d_population, h_population, sizeof(float) * populationSize * numCities, hipMemcpyHostToDevice);

    // Timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start, 0);

    for (int generation = 0; generation < numGenerations; ++generation) {
        calculateFitness<<<numBlocks, blockSize>>>(reinterpret_cast<int*>(d_population), d_fitness);
        // Declare and allocate memory for d_states
        hiprandState* d_states;
        hipMalloc((void**)&d_states, sizeof(hiprandState) * populationSize);

        // Call crossoverAndMutation function
        crossoverAndMutation<<<numBlocks, blockSize>>>(reinterpret_cast<int*>(d_population), d_fitness, d_states);
    }

    // Record stop time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Copy fitness results back to the host
    hipMemcpy(h_fitness, d_fitness, sizeof(float) * populationSize, hipMemcpyDeviceToHost);

    // Copy the population and fitness from device to host
    hipMemcpy(h_population, d_population, populationSize * numCities * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_fitness, d_fitness, populationSize * sizeof(float), hipMemcpyDeviceToHost);

    // Find the individual with the highest fitness
    int bestIndividual = 0;
    for (int i = 1; i < populationSize; ++i) {
        if (h_fitness[i] > h_fitness[bestIndividual]) {
            bestIndividual = i;
        }
    }

    // Print the best individual and its fitness
    printf("Best individual: ");
    for (int i = 0; i < numCities; ++i) {
        printf("%f ", h_population[bestIndividual * numCities + i]);
    }
    printf("\n");

    // Calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Time to " << numGenerations << " generations: " << elapsedTime << " ms\n";

    // Free device memory
    hipFree(d_population);
    hipFree(d_fitness);

    // Clean up host memory
    delete[] h_population;
    delete[] h_fitness;

    // Destroy timing events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
